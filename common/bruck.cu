#include "hip/hip_runtime.h"
// Source: https://github.com/harp-lab/rbruck_alltoall
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"
#include "nccl.h"

#include "error-catch.cu"
#include "typesize.cu"

int nccl_pow(int x, unsigned int p)
{
	if (p == 0)
	{
		return 1;
	}
	else if (p == 1)
	{
		return x;
	}

	int tmp = nccl_pow(x, p / 2);
	if (p % 2 == 0)
	{
		return tmp * tmp;
	}
	return x * tmp * tmp;
}

void ncclBruck(int r, char *d_send_data, int send_count, ncclDataType_t send_type, char *d_recv_data, int recv_count, ncclDataType_t recv_type, ncclComm_t comm, hipStream_t stream)
{
	int size;
	int rank;
	NCCLCHECK(ncclCommCount(comm, &size));
	NCCLCHECK(ncclCommUserRank(comm, &rank));

	if (r < 2 || size < 2)
	{
		std::cout << "Error: ncclBruck requires r >= 2 and nProc >= 2" << std::endl;
		return;
	}

	int unit_size = send_count * ncclTypeSize(send_type);
	int w = std::ceil(std::log(size) / std::log(r));
	int nlpow = nccl_pow(r, w - 1);
	int d = (nccl_pow(r, w) - size) / nlpow;

	CUDACHECK(hipMemcpy(d_recv_data, d_send_data, size * unit_size, hipMemcpyDefault));
	CUDACHECK(hipMemcpy(&d_send_data[(size - rank) * unit_size], d_recv_data, rank * unit_size, hipMemcpyDefault));
	CUDACHECK(hipMemcpy(d_send_data, &d_recv_data[rank * unit_size], (size - rank) * unit_size, hipMemcpyDefault));

	std::vector<std::vector<int>> rank_r_reps(size * w);
	for (int i = 0; i < size; i++)
	{
		std::vector<int> v(w);
		int N = i;
		int j = 0;
		while (N)
		{
			v[j++] = (N % r);
			N /= r;
		}
		rank_r_reps[i] = v;
	}

	int sent_blocks[nlpow];
	int di = 0;
	int ci = 0;

	char *temp_buffer;
	CUDACHECK(hipMalloc((void **)&temp_buffer, nlpow * unit_size));

	for (int x = 0; x < w; x++)
	{
		int ze = (x == w - 1) ? r - d : r;
		for (int z = 1; z < ze; z++)
		{
			di = 0;
			ci = 0;
			for (int i = 0; i < size; i++)
			{
				if (rank_r_reps[i][x] == z)
				{
					sent_blocks[di] = i;
					CUDACHECK(hipMemcpy(&temp_buffer[unit_size * ci], &d_send_data[unit_size * i], unit_size, hipMemcpyDefault));
					di += 1;
					ci += 1;
				}
			}

			int distance = z * nccl_pow(r, x);
			int recv_rank = (rank - distance + size) % size;
			int send_rank = (rank + distance) % size;

			NCCLCHECK(ncclGroupStart());
			NCCLCHECK(ncclSend(temp_buffer, di * unit_size, send_type, send_rank, comm, stream));
			NCCLCHECK(ncclRecv(d_recv_data, di * unit_size, recv_type, recv_rank, comm, stream));
			NCCLCHECK(ncclGroupEnd());

			for (int i = 0; i < di; i++)
			{
				CUDACHECK(hipMemcpy(&d_send_data[sent_blocks[i] * unit_size], &d_recv_data[i * unit_size], unit_size, hipMemcpyDefault));
			}
		}
	}

	for (int i = 0; i < size; i++)
	{
		CUDACHECK(hipMemcpy(&d_recv_data[((rank - i + size) % size) * unit_size], &d_send_data[i * unit_size], unit_size, hipMemcpyDefault));
	}

	CUDACHECK(hipFree(temp_buffer));
}
