// Source: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/usage/communicators.html

#include <hip/hip_runtime.h>
#include <sched.h>

#include "nccl.h"

int ncclStreamSynchronize(hipStream_t stream, ncclComm_t comm)
{
  hipError_t cudaErr;
  ncclResult_t ncclErr, ncclAsyncErr;
  while (1)
  {
    cudaErr = hipStreamQuery(stream);
    if (cudaErr == hipSuccess)
      return 0;

    if (cudaErr != hipErrorNotReady)
    {
      printf("CUDA Error : hipStreamQuery returned %d\n", cudaErr);
      return 1;
    }

    ncclErr = ncclCommGetAsyncError(comm, &ncclAsyncErr);
    if (ncclErr != ncclSuccess)
    {
      printf("NCCL Error : ncclCommGetAsyncError returned %d\n", ncclErr);
      return 1;
    }

    if (ncclAsyncErr != ncclSuccess)
    {
      // An asynchronous error happened. Stop the operation and destroy
      // the communicator
      ncclErr = ncclCommAbort(comm);
      if (ncclErr != ncclSuccess)
        printf("NCCL Error : ncclCommDestroy returned %d\n", ncclErr);
      // Caller may abort or try to re-create a new communicator.
      return 2;
    }

    // We might want to let other threads (including NCCL threads) use the CPU.
    sched_yield();
  }
}