#include "hip/hip_runtime.h"
// Source: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
#include <chrono>
#include <cstdint>
#include <fstream>
#include <iostream>
#include <numeric>
#include <vector>

#include <mpi.h>
#include <nccl.h>

#include "../common/error-catch.cpp"
#include "../common/error-catch.cu"
#include "../common/hostname.cu"

int main(int argc, char *argv[])
{
  // Initialize MPI
  MPICHECK(MPI_Init(&argc, &argv));

  // Set MPI size and rank
  int size;
  int rank;
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &size));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));

  int count;
  CUDACHECK(hipGetDeviceCount(&count));
  if (rank == 0)
  {
    std::cout << "nccl-ata" << std::endl;
    std::cout << "CUDA devices available: " << count << std::endl;
  }

  // Figure out what host the current MPI process is running on
  uint64_t hostHashs[size];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[rank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Compute and set the local rank based on the hostname
  int local_rank = 0;
  for (int i = 0; i < size; i++)
  {
    if (i == rank)
    {
      break;
    }
    if (hostHashs[i] == hostHashs[rank])
    {
      local_rank++;
    }
  }

  // Initialize a unique NCCL ID at process 0 and broadcast it to all others
  ncclUniqueId id;
  if (rank == 0)
  {
    ncclGetUniqueId(&id);
  }
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // Initialize NCCL
  ncclComm_t comm;
  hipStream_t stream;
  CUDACHECK(hipSetDevice(local_rank));
  CUDACHECK(hipStreamCreate(&stream));
  NCCLCHECK(ncclCommInitRank(&comm, size, id, rank));

  // Host variables
  int *h_send_data;
  int *h_recv_data;

  // Device variables
  int *d_send_data;
  int *d_recv_data;

  // Benchmark loop
  const int num_executions = 100;
  for (int i = 100; i <= 2000; i += 100)
  {
    // Send and recieve buffers must be the same size
    const int64_t buffer_size = size * i;
    const int64_t send_bytes = i * sizeof(int);
    const int64_t recv_bytes = buffer_size * sizeof(int);

    h_send_data = new int[i];
    h_recv_data = new int[buffer_size];
    CUDACHECK(hipMalloc((void **)&d_send_data, send_bytes));
    CUDACHECK(hipMalloc((void **)&d_recv_data, recv_bytes));

    // Fill the send buffer with each process rank
    for (int j = 0; j < i; j++)
    {
      h_send_data[j] = rank;
    }

    CUDACHECK(hipMemcpy(d_send_data, h_send_data, send_bytes, hipMemcpyDefault));
    CUDACHECK(hipMemset(d_recv_data, 0, recv_bytes));
    if (rank == 0)
    {
      std::cout << "Finished setting buffers" << std::endl;
    }

    // Warm-up loop
    for (int j = 0; j < 5; j++)
    {
      CUDACHECK(hipMemcpy(d_send_data, h_send_data, send_bytes, hipMemcpyDefault));
      CUDACHECK(hipMemset(d_recv_data, 0, recv_bytes));
      CUDACHECK(hipDeviceSynchronize());
      NCCLCHECK(ncclGroupStart());
      for (int k = 0; k < buffer_size; k++)
      {
        NCCLCHECK(ncclSend((void *)&d_send_data[k], i, ncclInt, k % size, comm, stream));
        NCCLCHECK(ncclRecv((void *)&d_recv_data[k], i, ncclInt, k % size, comm, stream));
      }
      NCCLCHECK(ncclGroupEnd());
    }

    if (rank == 0)
    {
      std::cout << "Finished warming up" << std::endl;
    }

    std::vector<double> times(num_executions);
    for (int j = 0; j < num_executions; j++)
    {
      // Reset buffers
      CUDACHECK(hipMemcpy(d_send_data, h_send_data, send_bytes, hipMemcpyDefault));
      CUDACHECK(hipMemset(d_recv_data, 0, recv_bytes));
      CUDACHECK(hipDeviceSynchronize());

      // Perform all-to-all
      auto start = std::chrono::high_resolution_clock::now();
      NCCLCHECK(ncclGroupStart());
      for (int k = 0; k < buffer_size; k++)
      {
        NCCLCHECK(ncclSend((void *)&d_send_data[k], i, ncclInt, k % size, comm, stream));
        NCCLCHECK(ncclRecv((void *)&d_recv_data[k], i, ncclInt, k % size, comm, stream));
      }
      NCCLCHECK(ncclGroupEnd());
      CUDACHECK(hipStreamSynchronize(stream));
      auto stop = std::chrono::high_resolution_clock::now();

      // Compute elapsed time
      auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
      const double localElapsedTime = duration.count();

      MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
      double elapsedTime;
      MPICHECK(MPI_Reduce(&localElapsedTime, &elapsedTime, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));
      if (rank == 0)
      {
        times[j] = localElapsedTime;
      }
    }

    if (rank == 0)
    {
      std::cout << "Finished benchmark loop" << std::endl;
    }

    MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
    if (rank == 0)
    {
      double sum = 0;
      for (int j = 0; j < num_executions; j++)
      {
        sum += times[j];
      }
      double average = sum / num_executions;

      std::ofstream log;
      log.open("run.log", std::ios_base::app);
      log << "nccl-ata w/ " << i * sizeof(int) << " bytes sent per GPU: " << average << " ns" << std::endl;
      log.close();

      std::cout << "Finished " << i * sizeof(int) << "-size byte benchmark" << std::endl;
    }

    // Verify that all ranks have the same thing in their recieve buffer
    // CUDACHECK(hipMemcpy(h_recv_data, d_recv_data, bytes, hipMemcpyDefault));
    // std::cout << "Rank " << rank << " received data: [";
    // for (int i = 0; i < buffer_size; i++) {
    //   std::cout << " " << h_recv_data[i] << " ";
    // }
    // std::cout << "]" << std::endl;

    // Free all allocated variables
    delete[] h_send_data;
    delete[] h_recv_data;
    CUDACHECK(hipFree(d_send_data));
    CUDACHECK(hipFree(d_recv_data));
  }

  // Destroy NCCL communicator
  ncclCommDestroy(comm);

  // Finalize MPI
  MPICHECK(MPI_Finalize());
  return 0;
}