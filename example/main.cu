// Source: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
#include <iostream>

#include <mpi.h>
#include <nccl.h>

#include "../common/bruck.cu"
#include "../common/hostname.cu"
#include "../common/typesize.cu"

int main(int argc, char *argv[])
{
  // Initialize MPI
  MPI_Init(&argc, &argv);

  // Set MPI size and rank
  int world_size;
  int mpi_rank;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);

  // Figure out what host the current MPI process is running on
  uint64_t hostHashs[world_size];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[mpi_rank] = getHostHash(hostname);
  MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD);

  // Compute and set the local rank based on the hostname
  int local_host_rank = 0;
  for (int i = 0; i < world_size; i++)
  {
    if (i == mpi_rank)
    {
      break;
    }
    if (hostHashs[i] == hostHashs[mpi_rank])
    {
      local_host_rank++;
    }
  }

  // Initialize a unique NCCL ID at process 0 and broadcast it to all others
  ncclUniqueId id;
  if (mpi_rank == 0)
  {
    ncclGetUniqueId(&id);
  }
  MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

  // Initialize NCCL
  ncclComm_t comm;
  hipStream_t stream;
  hipSetDevice(local_host_rank);
  hipStreamCreate(&stream);
  ncclCommInitRank(&comm, world_size, id, mpi_rank);

  // Host variables
  int *h_send_data;
  int *h_verify_data;
  int *h_recv_data;

  // Device variables
  int *d_send_data;
  int *d_recv_data;

  // Send and recieve buffers
  const int send_count = 1;
  const int buffer_size = send_count * world_size;
  const int buffer_bytes = buffer_size * sizeof(int);

  // Allocate host memory
  h_send_data = new int[buffer_size];
  h_verify_data = new int[buffer_size];
  h_recv_data = new int[buffer_size];

  // Allocate device memory
  hipMalloc((void **)&d_send_data, buffer_bytes);
  hipMalloc((void **)&d_recv_data, buffer_bytes);
  hipMemset(d_send_data, 0, buffer_bytes);
  hipMemset(d_recv_data, 0, buffer_bytes);

  // Prepare the send buffer
  for (int i = 0; i < buffer_size; i++)
  {
    h_send_data[i] = mpi_rank;
  }

  // Prepare the verification buffer
  for (int i = 0; i < world_size; i++)
  {
    for (int j = 0; j < send_count; j++)
    {
      h_verify_data[j + i * send_count] = i;
    }
  }

  // Copy host memory to device memory
  hipMemcpy(d_send_data, h_send_data, buffer_bytes, hipMemcpyHostToDevice);

  // NCCL all to all
  int nRanks;
  ncclCommCount(comm, &nRanks);
  const size_t rankOffset = send_count * ncclTypeSize(ncclInt);
  // ncclGroupStart();
  // for (int r = 0; r < nRanks; r++)
  // {
  //   ncclSend(((char *)d_send_data) + r * rankOffset, send_count, ncclInt, r, comm, stream);
  //   ncclRecv(((char *)d_recv_data) + r * rankOffset, send_count, ncclInt, r, comm, stream);
  // }
  // ncclGroupEnd();
  ncclBruck(2, (char *)d_send_data, send_count, ncclInt, (char *)d_recv_data, send_count, ncclInt, comm, stream);

  // Verify against the verification data
  hipMemcpy(h_recv_data, d_recv_data, buffer_bytes, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  std::cout << "Rank " << mpi_rank << ": received data: [";
  for (int j = 0; j < buffer_size; j++)
  {
    std::cout << " " << h_recv_data[j] << " ";
  }
  std::cout << "]" << std::endl;

  bool passed = true;
  for (int i = 0; i < buffer_size; i++)
  {
    if (h_recv_data[i] != h_verify_data[i])
    {
      passed = false;
    }
  }

  if (passed)
  {
    std::cout << "Rank " << mpi_rank << ": passed" << std::endl;
  }
  else
  {
    std::cout << "Rank " << mpi_rank << ": failed" << std::endl;
  }

  // Free all allocated memory
  delete[] h_recv_data;
  hipFree(d_send_data);
  hipFree(d_recv_data);

  // Destroy NCCL communicator
  ncclCommDestroy(comm);

  // Finalize MPI
  MPI_Finalize();
  return 0;
}