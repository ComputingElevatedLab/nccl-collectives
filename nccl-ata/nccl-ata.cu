#include "hip/hip_runtime.h"
// Source:
// https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
#include <fstream>
#include <iostream>
#include <numeric>
#include <vector>

#include <mpi.h>
#include <nccl.h>

#include "../common/error-catch.cpp"
#include "../common/error-catch.cu"
#include "../common/hostname.cu"

int main(int argc, char *argv[]) {
  // Initialize MPI
  MPICHECK(MPI_Init(&argc, &argv));

  // Set MPI size and rank
  int size;
  int rank;
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &size));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));

  int count;
  CUDACHECK(hipGetDeviceCount(&count));
  if (rank == 0) {
    std::cout << "nccl-ata" << std::endl;
    std::cout << "CUDA devices available: " << count << std::endl;
  }

  // Figure out what host the current MPI process is running on
  uint64_t hostHashs[size];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[rank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Compute and set the local rank based on the hostname
  int local_rank = 0;
  for (int i = 0; i < size; i++) {
    if (i == rank) {
      break;
    }
    if (hostHashs[i] == hostHashs[rank]) {
      local_rank++;
    }
  }

  // Initialize a unique NCCL ID at process 0 and broadcast it to all others
  ncclUniqueId id;
  if (rank == 0) {
    ncclGetUniqueId(&id);
  }
  MPICHECK(MPI_Bcast((void*) &id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // Initialize NCCL
  ncclComm_t comm;
  hipStream_t stream;
  CUDACHECK(hipSetDevice(local_rank));
  CUDACHECK(hipStreamCreate(&stream));
  NCCLCHECK(ncclCommInitRank(&comm, size, id, rank));

  // Host variables
  int *h_send_data;
  int *h_recv_data;

  // Device variables
  int *d_send_data;
  int *d_recv_data;

  // Benchmark loop
  const int num_executions = 100;
  for (int i = 1; i <= 1000; i *= 10) {
    // Send and recieve buffers must be the same size
    const int buffer_size = size * i;
    const int bytes = buffer_size * sizeof(int);

    h_send_data = new int[buffer_size];
    h_recv_data = new int[buffer_size];
    CUDACHECK(hipMalloc((void**) &d_send_data, bytes));
    CUDACHECK(hipMalloc((void**) &d_recv_data, bytes));

    // Fill the send buffer with each process rank
    for (int j = 0; j < buffer_size; j++) {
      h_send_data[j] = rank;
    }
    
    CUDACHECK(hipMemcpy(d_send_data, h_send_data, bytes, hipMemcpyDefault));
    CUDACHECK(hipMemset(d_recv_data, 0, bytes));
    if (rank == 0) {
      std::cout << "Finished setting buffers" << std::endl;
    }

    // Warm-up loop
    for (int j = 0; j < 5; j++) {
      CUDACHECK(hipMemcpy(d_send_data, h_send_data, bytes, hipMemcpyDefault));
      CUDACHECK(hipMemset(d_recv_data, 0, bytes));
      MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
      NCCLCHECK(ncclGroupStart());
      for (int k = 0; k < buffer_size; k++) {
        NCCLCHECK(ncclSend((void*) &d_send_data[k], i, ncclInt, k % size, comm, stream));
        NCCLCHECK(ncclRecv((void*) &d_recv_data[k], i, ncclInt, k % size, comm, stream));
      }
      NCCLCHECK(ncclGroupEnd());
    }

    if (rank == 0) {
      std::cout << "Finished warming up" << std::endl;
    }

    std::vector<float> times(num_executions);
    for (int j = 0; j < num_executions; j++) {
      // Reset buffers
      CUDACHECK(hipMemcpy(d_send_data, h_send_data, bytes, hipMemcpyDefault));
      CUDACHECK(hipMemset(d_recv_data, 0, bytes));
      MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

      // Create CUDA events
      hipEvent_t start, stop;
      CUDACHECK(hipEventCreate(&start));
      CUDACHECK(hipEventCreate(&stop));

      // Perform all-to-all
      hipEventRecord(start, 0);
      NCCLCHECK(ncclGroupStart());
      for (int k = 0; k < buffer_size; k++) {
        NCCLCHECK(ncclSend((void*) &d_send_data[k], i, ncclInt, k % size, comm, stream));
        NCCLCHECK(ncclRecv((void*) &d_recv_data[k], i, ncclInt, k % size, comm, stream));
      }
      NCCLCHECK(ncclGroupEnd());
      CUDACHECK(hipEventRecord(stop, 0));
      CUDACHECK(hipEventSynchronize(stop));

      // Compute elapsed time
      float localElapsedTime;
      CUDACHECK(hipEventElapsedTime(&localElapsedTime, start, stop));

      // Destroy CUDA events
      CUDACHECK(hipEventDestroy(start));
      CUDACHECK(hipEventDestroy(stop));

      MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
      float elapsedTime;
      MPICHECK(MPI_Reduce(&localElapsedTime, &elapsedTime, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD));
      if (rank == 0) {
        times[j] = localElapsedTime;
      }
    }

    if (rank == 0) {
      std::cout << "Finished benchmark loop" << std::endl;
    }

    MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
    if (rank == 0) {
      float sum = 0;
      for (int i = 0; i < num_executions; i++) {
        sum += times[i];
      }
      float average = sum / num_executions;

      std::ofstream log;
      log.open("run.log", std::ios_base::app);
      log << "nccl-ata w/ " << bytes << " byte buffer: " << average << " ms" << std::endl;
      log.close();

      std::cout << "Finished " << bytes << "-size buffer benchmark" << std::endl;
    }
    
    // Verify that all ranks have the same thing in their recieve buffer
    // CUDACHECK(hipMemcpy(h_recv_data, d_recv_data, bytes, hipMemcpyDefault));
    // std::cout << "Rank " << rank << " received data: [";
    // for (int i = 0; i < buffer_size; i++) {
    //   std::cout << " " << h_recv_data[i] << " ";
    // }
    // std::cout << "]" << std::endl;

    // Free all allocated variables
    delete[] h_send_data;
    delete[] h_recv_data;
    CUDACHECK(hipFree(d_send_data));
    CUDACHECK(hipFree(d_recv_data));
  }

  // Destroy NCCL communicator
  ncclCommDestroy(comm);

  // Finalize MPI
  MPICHECK(MPI_Finalize());
  return 0;
}