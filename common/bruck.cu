#include "hip/hip_runtime.h"
// Source: https://github.com/harp-lab/rbruck_alltoall
#include <cmath>
#include <cstring>
#include <vector>

#include "nccl.h"

#include "error-catch.cu"

int myPow(int x, unsigned int p) {
    if (p == 0) return 1;
    if (p == 1) return x;
    int tmp = myPow(x, p/2);
    if (p%2 == 0) return tmp * tmp;
    else return x * tmp * tmp;
}

std::vector<int> convert10tob(int w, int N, int b) {
	std::vector<int> v(w);
	int i = 0;
	while(N) {
	  v[i++] = (N % b);
	  N /= b;
	}
	return v;
}

void ncclBruck(int r, char* sendbuf, int sendcount, ncclDataType_t sendtype, char* recvbuf, int recvcount, ncclDataType_t recvtype,  ncclComm_t comm) {
    int rank, nprocs;

    // Find NCCL equivalent for this
    // MPI_Comm_rank(comm, &rank);
    // MPI_Comm_size(comm, &nprocs);

    int typesize = sizeof(sendtype);
    int unit_size = sendcount * typesize;
    int w = ceil(std::log(nprocs) / std::log(r)); // calculate the number of digits when using r-representation
	int nlpow = myPow(r, w-1);
	int d = (myPow(r, w) - nprocs) / nlpow; // calculate the number of highest digits

    // local rotation
    CUDA_CALL(hipMemcpy(recvbuf, sendbuf, nprocs * unit_size, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(&sendbuf[(nprocs - rank) * unit_size], recvbuf, rank * unit_size, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(sendbuf, &recvbuf[rank * unit_size], (nprocs - rank) * unit_size, hipMemcpyDeviceToDevice));

    // convert rank to base r representation
    int* rank_r_reps;
    CUDA_CALL(hipMalloc(&rank_r_reps, nprocs * w * sizeof(int)));
	for (int i = 0; i < nprocs; i++) {
		std::vector<int> r_rep = convert10tob(w, i, r);
		CUDA_CALL(hipMemcpy(&rank_r_reps[i * w], r_rep.data(), w * sizeof(int), hipMemcpyDeviceToDevice));
	}

	int sent_blocks[nlpow];
	int sent_blocks_comp[nlpow];
	int di = 0;
	int ci = 0;

	int comm_steps = (r - 1) * w - d;
	char* temp_buffer;
    CUDA_CALL(hipMalloc(&temp_buffer, nlpow * unit_size));

	// communication steps = (r - 1)w - d
    for (int x = 0; x < w; x++) {
    	int ze = (x == w - 1)? r - d: r;
    	for (int z = 1; z < ze; z++) {
    		// get the sent data-blocks
    		// copy blocks which need to be sent at this step
    		di = 0;
    		ci = 0;
    		for (int i = 0; i < nprocs; i++) {
    			if (rank_r_reps[i*w + x] == z) {
    				sent_blocks[di++] = i;
    				CUDA_CALL(hipMemcpy(&temp_buffer[unit_size*ci++], &sendbuf[i*unit_size], unit_size, hipMemcpyDeviceToDevice));
    			}
    		}

    		int distance = z * myPow(r, x); // pow(1, 51) = 51, int d = pow(1, 51); // 50
    		int recv_proc = (rank - distance + nprocs) % nprocs; // receive data from rank - 2^step process
    		int send_proc = (rank + distance) % nprocs; // send data from rank + 2^k process
    		long long comm_size = di * unit_size;

            hipStream_t stream;
            NCCL_CALL(ncclGroupStart());
            NCCL_CALL(ncclSend(temp_buffer, comm_size, ncclChar, send_proc, comm, stream));
            NCCL_CALL(ncclRecv(temp_buffer, recvcount, ncclChar, recv_proc, comm, stream));
            NCCL_CALL(ncclGroupEnd());

    		// replace with received data
    		for (int i = 0; i < di; i++) {
    			long long offset = sent_blocks[i] * unit_size;
    			CUDA_CALL(hipMemcpy(sendbuf + offset, recvbuf+(i*unit_size), unit_size, hipMemcpyDeviceToDevice));
    		}
    	}
    }

    hipFree(rank_r_reps);
    hipFree(temp_buffer);

    // local rotation
	for (int i = 0; i < nprocs; i++) {
		int index = (rank - i + nprocs) % nprocs;
		CUDA_CALL(hipMemcpy(&recvbuf[index*unit_size], &sendbuf[i*unit_size], unit_size, hipMemcpyDeviceToDevice));
	}
}