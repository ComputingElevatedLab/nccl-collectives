#include "hip/hip_runtime.h"
// Source: https://github.com/harp-lab/rbruck_alltoall
#include <cmath>
#include <cstring>
#include <vector>

#include <stdio.h>

#include "nccl.h"

#include "error-catch.cu"

int myPow(int x, unsigned int p) {
    if (p == 0) return 1;
    if (p == 1) return x;
    int tmp = myPow(x, p/2);
    if (p%2 == 0) return tmp * tmp;
    else return x * tmp * tmp;
}

int* convert10tob(int w, int N, int b) {
	int* v = (int *) malloc(w * sizeof(int));
	int i = 0;
	while(N) {
	  v[i++] = (N % b);
	  N /= b;
	}
	return v;
}

void ncclBruck(int r, char* sendbuf, int sendcount, ncclDataType_t sendtype, char* recvbuf, int recvcount, ncclDataType_t recvtype,  ncclComm_t comm, hipStream_t stream, int rank, int size) {
    int typesize = sizeof(sendtype);
    int unit_size = sendcount * typesize;

	int w = 1;
	if (size != 0 && r == 0) {
		int w = ceil(std::log(size) / std::log(r));
	}
	int nlpow = myPow(r, w-1);
	int d = (myPow(r, w) - size) / nlpow;

    CUDA_CALL(hipMemcpy(recvbuf, sendbuf, size * unit_size, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(&sendbuf[(size - rank) * unit_size], recvbuf, rank * unit_size, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(sendbuf, &recvbuf[rank * unit_size], (size - rank) * unit_size, hipMemcpyDeviceToDevice));

    int* rank_r_reps;
    CUDA_CALL(hipMalloc((void **) &rank_r_reps, size * w * sizeof(int)));
	for (int i = 0; i < size; i++) {
		// TODO: Probably better to turn this into a kernel
		int* r_rep = convert10tob(w, i, r);
		CUDA_CALL(hipMemcpy((void *) &rank_r_reps[i * w], r_rep, w * sizeof(int), hipMemcpyHostToDevice));
		free(r_rep);
	}

	int sent_blocks[nlpow];
	int di = 0;
	int ci = 0;

	char* temp_buffer;
    CUDA_CALL(hipMalloc((void **) &temp_buffer, nlpow * unit_size));

    for (int x = 0; x < w; x++) {
    	int ze = (x == w - 1)? r - d: r;
    	for (int z = 1; z < ze; z++) {
    		di = 0;
    		ci = 0;
    		for (int i = 0; i < size; i++) {
				// TODO: Not allowed to access device memory from host memory
    			if (rank_r_reps[i*w + x] == z) {
    				sent_blocks[di++] = i;
    				CUDA_CALL(hipMemcpy(&temp_buffer[unit_size*ci++], &sendbuf[i*unit_size], unit_size, hipMemcpyDeviceToDevice));
    			}
    		}

    		int distance = z * myPow(r, x);
    		int recv_proc = (rank - distance + size) % size;
    		int send_proc = (rank + distance) % size;
    		long long comm_size = di * unit_size;

            NCCL_CALL(ncclGroupStart());
            NCCL_CALL(ncclSend(temp_buffer, comm_size, ncclChar, send_proc, comm, stream));
            NCCL_CALL(ncclRecv(temp_buffer, recvcount, ncclChar, recv_proc, comm, stream));
            NCCL_CALL(ncclGroupEnd());

    		for (int i = 0; i < di; i++) {
    			long long offset = sent_blocks[i] * unit_size;
    			CUDA_CALL(hipMemcpy(sendbuf + offset, recvbuf+(i*unit_size), unit_size, hipMemcpyDeviceToDevice));
    		}
    	}
    }

    hipFree(rank_r_reps);
    hipFree(temp_buffer);

	for (int i = 0; i < size; i++) {
		int index = (rank - i + size) % size;
		CUDA_CALL(hipMemcpy(&recvbuf[index*unit_size], &sendbuf[i*unit_size], unit_size, hipMemcpyDeviceToDevice));
	}
}